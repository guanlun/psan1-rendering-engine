#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "helpers.h"
#include "common_structs.h"
#include "node_shading_system.h"
#include "random.h"

using namespace optix;

rtDeclareVariable(uint2, thread_index, rtLaunchIndex, );
rtDeclareVariable(uint2, thread_dim, rtLaunchDim, );

rtDeclareVariable(rtObject, top_object, , );
rtDeclareVariable(rtObject, top_shadower, , );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );

rtDeclareVariable(float, scene_epsilon, , );
rtDeclareVariable(int, max_depth, , );

rtDeclareVariable(unsigned int, radiance_ray_type, , );
rtDeclareVariable(unsigned int, shadow_ray_type, , );

rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(float3, tangent, attribute tangent, ); 
rtDeclareVariable(float3, bitangent, attribute bitangent, ); 

rtDeclareVariable(float3, front_hit_point, attribute front_hit_point, );
rtDeclareVariable(float3, back_hit_point, attribute back_hit_point, );

rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, isect_dist, rtIntersectionDistance, );

rtDeclareVariable(float3, ambient_light_color, , );

rtDeclareVariable(int, is_emissive, , );
rtDeclareVariable(float3, k_emission, , );

rtDeclareVariable(float3, k_diffuse, , );

rtDeclareVariable(float3, k_specular, , );

rtDeclareVariable(float3, k_reflective, , );
rtDeclareVariable(float, glossiness, , );

rtDeclareVariable(float3, alpha, , );
rtDeclareVariable(float, IOR, , ) = 1.4;

rtDeclareVariable(float3, texcoord, attribute texcoord, ); 
rtDeclareVariable(float3, cutoff_color, , );

rtDeclareVariable(float, importance_cutoff, , );

rtDeclareVariable(float3, emissive_color, ,);
rtDeclareVariable(float, emissive_weight, ,);

rtDeclareVariable(float3, diffuse_color, ,);
rtDeclareVariable(float, diffuse_weight, ,);

rtDeclareVariable(float3, reflective_color, ,);
rtDeclareVariable(float, reflective_weight, ,);

rtDeclareVariable(float3, refractive_color, ,);
rtDeclareVariable(float, refractive_weight, ,);

rtDeclareVariable(float3, subsurf_scatter_color, , );
rtDeclareVariable(float, subsurf_scatter_weight, , );
rtDeclareVariable(float, subsurf_att, , ) = 0.1;

rtDeclareVariable(int, has_diffuse_map, , );
rtDeclareVariable(int, has_normal_map, , );
rtDeclareVariable(int, has_specular_map, , );

rtDeclareVariable(int, anisotropic, , );

rtTextureSampler<float4, 2> kd_map = NULL;
rtTextureSampler<float4, 2> ks_map = NULL;
rtTextureSampler<float4, 2> normal_map = NULL;

rtDeclareVariable(int, soft_shadow_on, ,) = false;
rtDeclareVariable(int, glossy_on, ,) = false;
rtDeclareVariable(int, gi_on, ,) = false;

rtDeclareVariable(unsigned int, frame_number, , );

rtBuffer<RectangleLight> area_lights;
rtBuffer<SpotLight> spot_lights;
rtBuffer<DirectionalLight> directional_lights;

struct PerRayData_radiance {
	float3 result;
	float importance;
	int depth;
	int ss;
};

struct PerRayData_shadow {
	float3 attenuation;
};

rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(PerRayData_shadow, prd_shadow, rtPayload, );

static __device__ __inline__ float3 exp( const float3& x ) {
	return make_float3(exp(x.x), exp(x.y), exp(x.z));
}

/* vector can be ignored */
static __device__ __inline__ bool float_vec_zero(float3 v) {
	return length(v) < scene_epsilon;
}

/*	calculate fresnel reflection */
static __device__ __inline__ float3 schlick(float nDi, const float3& rgb) {
	float r = fresnel_schlick(nDi, 5, rgb.x, 1);
	float g = fresnel_schlick(nDi, 5, rgb.y, 1);
	float b = fresnel_schlick(nDi, 5, rgb.z, 1);
	return make_float3(r, g, b);
}

static __device__ __inline__ float float3_sum(float3 v) {
	return v.x + v.y + v.z;
}

/*  randomize a vector based on normal distribution, used for 
    normal vectors in glossy reflection and refraction */
static __device__ float3 randomize_vector(const float3& v, float amount, unsigned int& seed) {
	// two random number for normal distribution
	float rand1 = rnd(seed), rand2 = rnd(seed);

	// X and Y are normally distributed random numbers
	float X = sqrt(- 2 * log(rand1)) * cos(2 * 3.14159 * rand2) * amount / 5;
	float Y = sqrt(- 2 * log(rand1)) * sin(2 * 3.14159 * rand2) * amount / 5;

	// make a vector not parallel to v to find v's tangent and bitangent
	float3 u = v;
	u.x += 1;

	// tangent
	float3 e1 = cross(u, v);

	//bitangent
	float3 e2 = cross(v, e1);

	normalize(e1);
	normalize(e2);

	// randomize v in its tangent and bitangent direction
	float3 rand_vec = v + e1 * X + e2 * Y;
	normalize(rand_vec);

	return rand_vec;
}

static __device__ __inline__ void createONB( const optix::float3& n, optix::float3& U, optix::float3& V) {
	U = cross( n, make_float3( 0.0f, 1.0f, 0.0f ) );
	if ( dot(U, U) < 1.e-3f ) {
		U = cross( n, make_float3( 1.0f, 0.0f, 0.0f ) );
	}
	U = normalize( U );
	V = cross( n, U );
}

static __device__ __inline__ float3 TraceRay(float3 origin, float3 direction, int depth, 
											 float importance, int ss = false) {
	optix::Ray ray = optix::make_Ray( origin, direction, radiance_ray_type, 0.0f, RT_DEFAULT_MAX );
	PerRayData_radiance prd;
	prd.result = make_float3(0);
	prd.depth = depth;
	prd.importance = importance;
	prd.ss = ss;

	rtTrace( top_object, ray, prd );
	return prd.result;
}

RT_PROGRAM void any_hit_shadow()
{
	if (!is_emissive) { // the hit object is not emissive
		prd_shadow.attenuation *= alpha;

		if (float_vec_zero(prd_shadow.attenuation)) {
			rtTerminateRay();
		} else {
			rtIgnoreIntersection();
		}
	}
	/*
	float3 world_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
  float nDi = fabs(dot(world_normal, ray.direction));

  prd_shadow.attenuation *= 1-fresnel_schlick(nDi, 5, 1-make_float3(1), make_float3(1));
  if(optix::luminance(prd_shadow.attenuation) < importance_cutoff)
    rtTerminateRay();
  else
    rtIgnoreIntersection();
	*/
}

RT_PROGRAM void closest_hit_radiance()
{
	// seed used for random number generation
	unsigned int seed = tea<16>(thread_index.y * thread_dim.x + thread_index.x, thread_index.y + frame_number);

	float3 kd;
	if (has_diffuse_map) { // has diffuse map, sample the texture
		kd = make_float3( tex2D( kd_map, texcoord.x, texcoord.y ) );
	} else {
		kd = k_diffuse;
	}

	// front hit point
	float3 fhp = rtTransformPoint(RT_OBJECT_TO_WORLD, front_hit_point);

	// back hit point
	float3 bhp = rtTransformPoint(RT_OBJECT_TO_WORLD, back_hit_point);

	// a trick to replace the diffuse factor with ss color
	if (prd_radiance.ss) {
		kd = subsurf_scatter_color;
		fhp = bhp;
	}

	// Here tangent, bitangent and normal are attribute variables set in the ray-generation program,
	// transform them to the work space using the normal transformation matrix
	const float3 T = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, tangent)); // tangent	
	const float3 B = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, bitangent)); // bitangent	
	const float3 N = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal)); // normal	

	// normal vector after considering normal map (if any)
	float3 normal;

	if (has_normal_map) { // has normal map, sample the texture
		const float3 k_normal = make_float3( tex2D( normal_map, texcoord.x, texcoord.y) );
		const float3 coeff = k_normal * 2 - make_float3(1); // transform from RGB to normal
		normal = T * coeff.x + B * coeff.y + N * coeff.z;
	} else {
		normal = N;
	}

	prd_radiance.result = k_emission;

	float3 diffuse_result = make_float3(0);
		
	for (int i = 0; i < area_lights.size(); i++) {
		RectangleLight light = area_lights[i];
		float3 sampledPos;

		if (soft_shadow_on) {
			sampledPos = light.pos + rnd(seed) * light.r1 + rnd(seed) * light.r2;
		} else {
			sampledPos = light.pos + 0.5 * light.r1 + 0.5 * light.r2;
		}

		float Ldist = length(sampledPos - fhp);

		float distance_attenuation = 1 / (1 + light.attenuation_coeff * Ldist 
			+ light.attenuation_coeff * Ldist * Ldist);

		float3 L = normalize(sampledPos - fhp);
		float3 H = normalize(L - ray.direction);

		float nDl = dot(normal, L);

		// cast shadow ray
		PerRayData_shadow shadow_prd;
		shadow_prd.attenuation = make_float3(1);

		if(nDl > 0) {
			optix::Ray shadow_ray = optix::make_Ray( fhp, L, shadow_ray_type, scene_epsilon, Ldist );
			rtTrace(top_shadower, shadow_ray, shadow_prd);

			diffuse_result += light.intensity * light.color * distance_attenuation * shadow_prd.attenuation 
				* (kd * nDl);
		}
	}

	for (int i = 0; i < spot_lights.size(); i++) {
		SpotLight light = spot_lights[i];

		float Ldist = length(light.pos - fhp);

		float distance_attenuation = 1 / (1 + light.attenuation_coeff * Ldist 
			+ light.attenuation_coeff * Ldist * Ldist);

		float3 L = normalize(light.pos - fhp);
		float3 H = normalize(L - ray.direction);

		float nDl = dot(normal, L);

		// different in direction
		float dir_diff = dot(normalize(light.direction), -L);

		// intensity of spotlight drops as the angle increases
		float angle_attenuation = pow(dir_diff, light.dropoff_rate);

		PerRayData_shadow shadow_prd;
		shadow_prd.attenuation = make_float3(1);

		if(nDl > 0 && dir_diff > cos(light.angle)) {
			optix::Ray shadow_ray = optix::make_Ray( fhp, L, shadow_ray_type, scene_epsilon, Ldist );
			rtTrace(top_shadower, shadow_ray, shadow_prd);
			diffuse_result += angle_attenuation * light.intensity * light.color 
				* distance_attenuation * shadow_prd.attenuation 
				* (kd * nDl);
		}
	}

	for (int i = 0; i < directional_lights.size(); i++) {
		DirectionalLight light = directional_lights[i];

		float3 L = normalize(- light.direction);
		float3 H = normalize(L - ray.direction);

		float nDl = dot(normal, L);

		PerRayData_shadow shadow_prd;
		shadow_prd.attenuation = make_float3(1);

		if (nDl > 0) {
			optix::Ray shadow_ray = optix::make_Ray( fhp, L, shadow_ray_type, scene_epsilon, RT_DEFAULT_MAX );
			rtTrace(top_shadower, shadow_ray, shadow_prd);
			diffuse_result += light.intensity * light.color * shadow_prd.attenuation
				* (kd * nDl);
		}
	}


	const int new_depth = prd_radiance.depth + 1;

	if (new_depth > max_depth) { // max depth exceeded, stop further tracing
		return;
	}

	float diffuse_amount = length(kd);
	float reflective_amount = length(k_reflective);
	float refractive_amount = length(alpha);
	float total_amount = diffuse_amount + reflective_amount + refractive_amount;

	/* global illunimation */
	float diffuse_importance = diffuse_amount / total_amount * prd_radiance.importance;

	if (gi_on && diffuse_importance > importance_cutoff) {
		// randomly sample a vector in the hemisphere 
		float3 p;
		cosine_sample_hemisphere(rnd(seed), rnd(seed), p);

		// create two vectors perpendicular to the normal
		float3 v1, v2;
		createONB(normal, v1, v2);

		float3 random_ray_direction = v1 * p.x + v2 * p.y + normal * p.z;

		// reduce importance more for indirect lighting to speed up rendering (divide by 2)
		diffuse_result += 0.8 * diffuse_importance * kd 
			* TraceRay(fhp, random_ray_direction, new_depth, diffuse_importance / 2);
	}
	
	// stop here if the ray is from subsurface scattering
	if (prd_radiance.ss) {
		float distance = length(fhp - ray.origin);
		float attenuation = 1 / (1 + subsurf_att * distance + subsurf_att * distance * distance);
		prd_radiance.result = diffuse_result * attenuation;
		return;
	}

	/* refraction */
	float fresnel_reflection = 1;
	float refractive_importance = refractive_amount / total_amount * prd_radiance.importance;

	if (!float_vec_zero(alpha) && refractive_importance > importance_cutoff) {

		float3 transmission_direction;
		if (refract(transmission_direction, ray.direction, normal, IOR)) {
			// check whether it is internal or external refraction
			float cos_theta = dot(ray.direction, normal);
			if (cos_theta < 0) { // external
				cos_theta = - cos_theta;
			} else { // internal
				cos_theta = dot(transmission_direction, normal);
			}

			fresnel_reflection = fresnel_schlick(cos_theta, 3, 0.1, 1);

			float importance = prd_radiance.importance * (1.0f - fresnel_reflection) * luminance(alpha);

			if (importance > importance_cutoff) {
				if (glossy_on) {
					float3 randomizedRefr = randomize_vector(transmission_direction, glossiness, seed);
					prd_radiance.result += (1 - fresnel_reflection) * alpha
						* TraceRay(bhp, randomizedRefr, new_depth, importance);
				} else {
					prd_radiance.result += (1 - fresnel_reflection) * alpha 
						* TraceRay(bhp, transmission_direction, new_depth, importance);
				}
			}
		}
	}

	/* reflection */
	float3 glossy_result = make_float3(0);
	// float reflective_importance = reflective_amount / total_amount * prd_radiance.importance;

	if (!float_vec_zero(k_reflective)) {
		// reflection direction
		const float3 refl = reflect(ray.direction, normal);

		float importance = prd_radiance.importance * fresnel_reflection * luminance(k_reflective);

		if (importance > importance_cutoff) {
			if (glossy_on) {
				float3 randomizedRefl;
				if (anisotropic) {
					if (rnd(seed) > 0.0) {
						randomizedRefl = normalize(refl + B * (rnd(seed) - 0.5) * 2);
					} else {
						randomizedRefl = randomize_vector(refl, glossiness, seed);
					}
				} else {
					randomizedRefl = randomize_vector(refl, glossiness, seed);
				}
				glossy_result += fresnel_reflection * k_reflective
					* TraceRay(fhp, randomizedRefl, new_depth, importance);
			} else {
				glossy_result += fresnel_reflection * k_reflective 
					* TraceRay(fhp, refl, new_depth, importance);
			}
		}
	}
	
	if (has_specular_map) {
		float3 ks = make_float3(tex2D(ks_map, texcoord.x, texcoord.y));
		float fac = (ks.x + ks.y + ks.z ) / 3.0;
		prd_radiance.result += (diffuse_result * (1 - fac) + glossy_result * fac) * 2;
	} else {
		prd_radiance.result += diffuse_result + glossy_result;
	}

	/* subsurface scattering */
	if (!float_vec_zero(subsurf_scatter_color)) {
		// randomly sample a vector in the hemisphere 
		float3 p;
		cosine_sample_hemisphere(rnd(seed), rnd(seed), p);

		// create two vectors perpendicular to the normal
		float3 v1, v2;
		createONB(normal, v1, v2);

		float3 random_ray_direction = - (v1 * p.x + v2 * p.y + normal * p.z);

		const float3 bhp = rtTransformPoint(RT_OBJECT_TO_WORLD, back_hit_point);

		prd_radiance.result += subsurf_scatter_color
			* TraceRay(bhp, random_ray_direction, new_depth, prd_radiance.importance * 0.8, true);
	}
}