#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "helpers.h"
#include "common_structs.h"
#include "random.h"

using namespace optix;

rtDeclareVariable(uint2, thread_index, rtLaunchIndex, );
rtDeclareVariable(uint2, thread_dim, rtLaunchDim, );

rtDeclareVariable(rtObject, top_object, , );
rtDeclareVariable(rtObject, top_shadower, , );
rtDeclareVariable(float, scene_epsilon, , );
rtDeclareVariable(int, max_depth, , );
rtDeclareVariable(unsigned int, radiance_ray_type, , );
rtDeclareVariable(unsigned int, shadow_ray_type, , );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(float3, tangent, attribute tangent, ); 
rtDeclareVariable(float3, bitangent, attribute bitangent, ); 
rtDeclareVariable(float3, front_hit_point, attribute front_hit_point, );
rtDeclareVariable(float3, back_hit_point, attribute back_hit_point, );

rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, isect_dist, rtIntersectionDistance, );

rtDeclareVariable(float3, ambient_light_color, , );

rtDeclareVariable(int, is_emissive, , );

rtDeclareVariable(float3, k_emission, , );
rtDeclareVariable(float3, k_ambient, , );
rtDeclareVariable(float3, k_diffuse, , );
rtDeclareVariable(float3, k_specular, , );
rtDeclareVariable(float3, k_reflective, , );
rtDeclareVariable(float3, k_refractive, , );
rtDeclareVariable(int, ns, , );
rtDeclareVariable(float, glossiness, , );

rtDeclareVariable(float3, texcoord, attribute texcoord, ); 
rtDeclareVariable(float3, cutoff_color, , );
rtDeclareVariable(int, reflection_maxdepth, , ) = 5;
rtDeclareVariable(int, refraction_maxdepth, , ) = 5;
rtDeclareVariable(float, importance_cutoff, , );
rtDeclareVariable(float, IOR, , ) = 1.3;

rtDeclareVariable(int, has_diffuse_map, , );
rtDeclareVariable(int, has_normal_map, , );
rtDeclareVariable(int, has_specular_map, , );

rtTextureSampler<float4, 2> kd_map = NULL;
rtTextureSampler<float4, 2> ks_map = NULL;
rtTextureSampler<float4, 2> normal_map = NULL;

rtDeclareVariable(int, soft_shadow_on, ,) = false;
rtDeclareVariable(int, glossy_on, ,) = false;
rtDeclareVariable(int, gi_on, ,) = false;

rtDeclareVariable(unsigned int, frame_number, , );

// rtBuffer<BasicLight> lights;
rtBuffer<RectangleLight> area_lights;

struct PerRayData_radiance
{
	float3 result;
	float importance;
	int depth;
};

struct PerRayData_shadow
{
	float3 attenuation;
};

rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(PerRayData_shadow, prd_shadow, rtPayload, );

#define PI 3.1415926

static __device__ __inline__ float float_vec_length(float3 v) {
	return sqrt(v.x * v.x + v.y * v.y + v.z * v.z);
}

static __device__ __inline__ bool float_vec_zero(float3 v) {
	return v.x < scene_epsilon && v.y < scene_epsilon && v.z < scene_epsilon;
}

/*	calculate fresnel reflection */
static __device__ __inline__ float3 schlick(float nDi, const float3& rgb) {
	float r = fresnel_schlick(nDi, 5, rgb.x, 1);
	float g = fresnel_schlick(nDi, 5, rgb.y, 1);
	float b = fresnel_schlick(nDi, 5, rgb.z, 1);
	return make_float3(r, g, b);
}

static __device__ __inline__ float float3_sum(float3 v) {
	return v.x + v.y + v.z;
}

/*  randomize a vector based on normal distribution, used for 
    normal vectors in glossy reflection and refraction */
static __device__ float3 randomize_vector(const float3& v, float amount, unsigned int& seed) {
	// two random number for normal distribution
	float rand1 = rnd(seed), rand2 = rnd(seed);

	// X and Y are normally distributed random numbers
	float X = sqrt(- 2 * log(rand1)) * cos(2 * PI * rand2) * amount / 5;
	float Y = sqrt(- 2 * log(rand1)) * sin(2 * PI * rand2) * amount / 5;

	// make a vector not parallel to v to find v's tangent and bitangent
	float3 u = v;
	u.x += 1;

	// tangent
	float3 e1 = cross(u, v);

	//bitangent
	float3 e2 = cross(v, e1);

	normalize(e1);
	normalize(e2);

	// randomize v in its tangent and bitangent direction
	float3 rand_vec = v + e1 * X + e2 * Y;
	normalize(rand_vec);

	return rand_vec;
}

static __device__ __inline__ void createONB( const optix::float3& n, optix::float3& U, optix::float3& V) {
	U = cross( n, make_float3( 0.0f, 1.0f, 0.0f ) );
	if ( dot(U, U) < 1.e-3f ) {
		U = cross( n, make_float3( 1.0f, 0.0f, 0.0f ) );
	}
	U = normalize( U );
	V = cross( n, U );
}

static __device__ __inline__ float3 TraceRay(float3 origin, float3 direction, int depth, float importance) {
	optix::Ray ray = optix::make_Ray( origin, direction, radiance_ray_type, 0.0f, RT_DEFAULT_MAX );
	PerRayData_radiance prd;
	prd.result = make_float3(0);
	prd.depth = depth;
	prd.importance = importance;

	rtTrace( top_object, ray, prd );
	return prd.result;
}

RT_PROGRAM void any_hit_shadow()
{
	if (!is_emissive) {
		prd_shadow.attenuation = make_float3(0.0); // TODO
		rtTerminateRay();
	}
}

RT_PROGRAM void closest_hit_radiance()
{
	/*
	if (prd_radiance.depth > max_depth || prd_radiance.importance < importance_cutoff) {
		prd_radiance.result = make_float3(0);
		return;
	}
	*/

	if (is_emissive) { // emissive object, just return the light color
		prd_radiance.result = k_emission;
		return;
	}

	// seed used for random number generation
	unsigned int seed = tea<16>(thread_index.y * thread_dim.x + thread_index.x, thread_index.y + frame_number);

	const float3 ray_dir = ray.direction; 

	float3 kd;
	if (has_diffuse_map) { // has diffuse map, sample the texture
		kd = make_float3( tex2D( kd_map, texcoord.x, texcoord.y ) );
	} else {
		kd = k_diffuse;
	}

	float3 ks;
	if (has_specular_map) { // has specular map, sample the texture
		ks = make_float3( tex2D( ks_map, texcoord.x, texcoord.y ) );
	} else {
		ks = k_specular;
	}

	// Here tangent, bitangent and normal are attribute variables set in the ray-generation program,
	// transform them to the work space using the normal transformation matrix
	const float3 T = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, tangent)); // tangent	
	const float3 B = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, bitangent)); // bitangent	
	const float3 N = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal)); // normal	

	// normal vector after considering normal map (if any)
	float3 normal;

	if (has_normal_map) { // has normal map, sample the texture
		const float3 k_normal = make_float3( tex2D( normal_map, texcoord.x, texcoord.y) );
		const float3 coeff = k_normal * 2 - make_float3(1, 1, 1); // transform from RGB to normal
		normal = T * coeff.x + B * coeff.y + N * coeff.z;
	} else {
		normal = N;
	}

	// first hit point
	const float3 fhp = rtTransformPoint(RT_OBJECT_TO_WORLD, front_hit_point);
	const float3 bhp = rtTransformPoint(RT_OBJECT_TO_WORLD, back_hit_point);

	// starting from the ambient color
	prd_radiance.result = kd * ambient_light_color;
		
	for (int i = 0; i < area_lights.size(); i++) {
		RectangleLight light = area_lights[i];

		// according to whether offline rendering is activated, use different number of samples to
		// achieve soft or hard shadow
		const int numShadowSamples = soft_shadow_on ? 10 : 1;

		for (int j = 0; j < numShadowSamples; j++) {
			float3 sampledPos;
			if (soft_shadow_on) {
				sampledPos = light.pos + rnd(seed) * light.r1 + rnd(seed) * light.r2;
			} else {
				sampledPos = light.pos + 0.5 * light.r1 + 0.5 * light.r2;
			}

			float Ldist = length(sampledPos - fhp);

			float3 L = normalize(sampledPos - fhp);
			float3 H = normalize(L - ray.direction);

			float nDl = dot(normal, L);

			// cast shadow ray
			PerRayData_shadow shadow_prd;
			shadow_prd.attenuation = make_float3(1);

			if(nDl > 0) {
				optix::Ray shadow_ray = optix::make_Ray( fhp, L, shadow_ray_type, scene_epsilon, Ldist );
				rtTrace(top_shadower, shadow_ray, shadow_prd);
				prd_radiance.result += light.color * shadow_prd.attenuation 
					* (kd * nDl + ks * max(pow(dot(H, normal), ns), .0f)) / numShadowSamples;
			}
		}
	}

	const int new_depth = prd_radiance.depth + 1;

	if (new_depth > max_depth) {
		return;
	}

	float ambient_amount = float_vec_length(k_ambient);
	float diffuse_amount = float_vec_length(k_diffuse);
	float specular_amount = float_vec_length(k_specular);
	float reflective_amount = float_vec_length(k_reflective);
	float refractive_amount = float_vec_length(k_refractive);
	float total_amount = ambient_amount + diffuse_amount 
		+ specular_amount + reflective_amount + refractive_amount;

	/* global illunimation */
	float diffuse_importance = diffuse_amount / total_amount * prd_radiance.importance;

	if (gi_on && diffuse_importance > importance_cutoff) {
		// randomly sample a vector in the hemisphere 
		float3 p;
		cosine_sample_hemisphere(rnd(seed), rnd(seed), p);

		// create two vectors perpendicular to the normal
		float3 v1, v2;
		createONB(normal, v1, v2);

		float3 random_ray_direction = v1 * p.x + v2 * p.y + normal * p.z;

		// reduce importance more for indirect lighting to speed up rendering (divide by 2)
		prd_radiance.result += diffuse_importance * kd 
			* TraceRay(fhp, random_ray_direction, new_depth, diffuse_importance / 2);
	}

	/* reflection */
	float reflective_importance = reflective_amount / total_amount * prd_radiance.importance;

	if (!float_vec_zero(k_reflective) && reflective_importance > importance_cutoff) {

		// reflection direction
		const float3 refl = reflect(ray_dir, normal);

		// number of samples to take for each reflection
		if (glossy_on) {
			const int num_glossy_sample = 10;
			
			for (int i = 0; i < num_glossy_sample; i++) {
				float3 randomizedRefl = randomize_vector(refl, glossiness, seed);
				prd_radiance.result += reflective_importance * k_reflective
					* TraceRay(fhp, randomizedRefl, new_depth, reflective_importance) / num_glossy_sample;
			}
		} else {
			prd_radiance.result += reflective_importance * k_reflective 
				* TraceRay(fhp, refl, new_depth, reflective_importance);
		}
	}

	/* refraction */
	float refractive_importance = refractive_amount / total_amount * prd_radiance.importance;

	if (!float_vec_zero(k_refractive) && refractive_importance > importance_cutoff) {

		float3 transmission_direction;
		if (refract(transmission_direction, ray_dir, N, IOR)) {
			// check whether it is internal or external refraction
			float cos_theta = dot(ray_dir, normal);
			if (cos_theta < 0) { // external
				cos_theta = - cos_theta;
			} else { // internal
				cos_theta = dot(transmission_direction, normal);
			}

			prd_radiance.result += refractive_importance * k_refractive 
				* TraceRay(bhp, transmission_direction, new_depth, refractive_importance / 2);
		}
	}
	
}

