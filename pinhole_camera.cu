#include "hip/hip_runtime.h"

/*
 * Copyright (c) 2008 - 2009 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and proprietary
 * rights in and to this software, related documentation and any modifications thereto.
 * Any use, reproduction, disclosure or distribution of this software and related
 * documentation without an express license agreement from NVIDIA Corporation is strictly
 * prohibited.
 *
 * TO THE MAXIMUM EXTENT PERMITTED BY APPLICABLE LAW, THIS SOFTWARE IS PROVIDED *AS IS*
 * AND NVIDIA AND ITS SUPPLIERS DISCLAIM ALL WARRANTIES, EITHER EXPRESS OR IMPLIED,
 * INCLUDING, BUT NOT LIMITED TO, IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A
 * PARTICULAR PURPOSE.  IN NO EVENT SHALL NVIDIA OR ITS SUPPLIERS BE LIABLE FOR ANY
 * SPECIAL, INCIDENTAL, INDIRECT, OR CONSEQUENTIAL DAMAGES WHATSOEVER (INCLUDING, WITHOUT
 * LIMITATION, DAMAGES FOR LOSS OF BUSINESS PROFITS, BUSINESS INTERRUPTION, LOSS OF
 * BUSINESS INFORMATION, OR ANY OTHER PECUNIARY LOSS) ARISING OUT OF THE USE OF OR
 * INABILITY TO USE THIS SOFTWARE, EVEN IF NVIDIA HAS BEEN ADVISED OF THE POSSIBILITY OF
 * SUCH DAMAGES
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>

#include "helpers.h"

using namespace optix;

struct PerRayData_radiance
{
  float3 result;
  float importance;
  int depth;
};

rtDeclareVariable(float3,        eye, , ) = { 1.0f, 0.0f, 0.0f };
rtDeclareVariable(float3,        U, , )   = { 0.0f, 1.0f, 0.0f };
rtDeclareVariable(float3,        V, , )   = { 0.0f, 0.0f, 1.0f };
rtDeclareVariable(float3,        W, , )   = { -1.0f, 0.0f, 0.0f };
rtDeclareVariable(float3,        bad_color, , );
rtDeclareVariable(float,         scene_epsilon, , ) = 0.1f;
rtDeclareVariable(rtObject,      top_object, , );
rtDeclareVariable(unsigned int,  radiance_ray_type, , );

rtBuffer<float4, 2>              output_buffer;

rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint2, launch_dim,   rtLaunchDim, );
rtDeclareVariable(float, time_view_scale, , ) = 1e-6f;

rtDeclareVariable(float, aperture_radius, , );
rtDeclareVariable(float, focal_scale, , );


// #define TIME_VIEW


__device__ __forceinline__ void write_output( float3 c )
{
  output_buffer[launch_index] = make_float4(c, 1.f);
}

__device__ __forceinline__ float3 read_output()
{
  return make_float3( output_buffer[launch_index] );
}

RT_PROGRAM void pinhole_camera()
{
#ifdef TIME_VIEW
  clock_t t0 = clock(); 
#endif
  float2 d = make_float2(launch_index) / make_float2(launch_dim) * 2.f - 1.f;
  float3 ray_origin = eye;
  float3 ray_direction = normalize(d.x*U + d.y*V + W);
  
  optix::Ray ray = optix::make_Ray(ray_origin, ray_direction, radiance_ray_type, scene_epsilon, RT_DEFAULT_MAX);

  PerRayData_radiance prd;
  prd.importance = 1.f;
  prd.depth = 0;

  rtTrace(top_object, ray, prd);

#ifdef TIME_VIEW
  clock_t t1 = clock(); 
 
  float expected_fps   = 1.0f;
  float pixel_time     = ( t1 - t0 ) * time_view_scale * expected_fps;
  write_output( make_float3( pixel_time ) );
#else
    write_output(prd.result);
#endif
}

RT_PROGRAM void exception()
{
  write_output(bad_color);
}
